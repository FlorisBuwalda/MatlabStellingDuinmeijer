#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <iostream>
#include <thread>
#include <mutex>
#include <cassert>
#include <condition_variable>
#include <chrono>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#define _POSIX_C_SOURCE 199309L
#include <time.h>
#include "hip/hip_runtime.h"
#define NOMINMAX
#define WIN32_LEAN_AND_MEAN
 /* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <windows.h>
#define CLOCK_REALTIME 0
//struct timespec { long tv_sec; long tv_nsec; };    //header par

#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <cyclicbarrier.hpp>

#ifndef __HIPCC__  
#define __HIPCC__
#endif
using namespace std;

 // Convenience function for checking CUDA runtime API results
 // can be wrapped around any runtime API call. No-op in release builds.
inline

hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
 		assert(result == hipSuccess);
	}
#endif
	return result;
}

////// parameters
constexpr int BLOCK_SIZE_x = 32;  // number of threads per block in x-dir
constexpr int BLOCK_SIZE_y = 32;  // number of threads per block in y-dir

//constexpr int threads[] = { 1,0 };
//constexpr int iter = 10;
//constexpr int ni = 1;
//constexpr int n = 3 * 32 * ni + 2;
//constexpr int L = n + 1;          //Domain length
//constexpr int W = n + 1;          //Domain width
//constexpr float dx_h = (float)W / ((float)(n + 1));                // inter grid distance in x - direction
//constexpr float dy_h = (float)L / ((float)(n + 1));                // inter grid distance in y - direction

int n;
constexpr float dx_h = 1;
constexpr float dy_h = 1;


__constant__ int  n_d;
__constant__ float dx;
__constant__ float dy;

__constant__ float Hstart ;    //Rest water depth
constexpr float Hstart_h = 1;

constexpr float g_h = (float)9.8;   // gravitational constant
__constant__ float g;                

constexpr float tstep = 1;             // maximum timestep
constexpr float dt = (float).01;                // first step is maximum timestep





__constant__ float cf;
constexpr float cf_h = 0;                    // Bottom friction factor

constexpr float droppar_h = .5;
__constant__ float droppar;

constexpr int ndrops = 1;              // maximum number of water drops
constexpr int dropstep = 5;            // drop interval

bool timer = false;
constexpr float safety = (float).9;



	__global__ void update( float *h, __int8 *upos, __int8 *vpos, float *U, float *V, float dt )
{
	
		
		__shared__      float s_h[BLOCK_SIZE_y+2][BLOCK_SIZE_x+2]; // 4-wide halo
		//__shared__     float s_hy[BLOCK_SIZE_y+2][BLOCK_SIZE_x+2]; // 4-wide halo
		//__shared__     float s_hx[BLOCK_SIZE_y+2][BLOCK_SIZE_x+2]; // 4-wide halo
		__shared__  __int8 s_upos[BLOCK_SIZE_y+2][BLOCK_SIZE_x+2]; // 2-wide halo
		__shared__  __int8 s_vpos[BLOCK_SIZE_y+2][BLOCK_SIZE_x+2]; // 2-wide halo
		__shared__      float s_U[BLOCK_SIZE_y+2][BLOCK_SIZE_x+2]; // 2-wide halo
		__shared__      float s_V[BLOCK_SIZE_y+2][BLOCK_SIZE_x+2]; // 2-wide halo

		//int i = threadIdx.x;
		//int j = blockIdx.x*blockDim.y + threadIdx.y;
		int j = blockIdx.y*blockDim.y + threadIdx.y;
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		int si = threadIdx.x + 1; // local i for shared memory access + halo offset
		int sj = threadIdx.y + 1; // local j for shared memory access
		float utemp, vtemp;
		float s_hx, s_hxmin, s_hy, s_hymin;
		int globalIdx =  (j+1) * n_d + i+1;
		
		//Boundaries
		if (threadIdx.x ==0) {
			
			   s_h[sj][si-1] =    h[globalIdx-1];
			s_upos[sj][si-1] = upos[globalIdx-1];
			s_vpos[sj][si-1] = vpos[globalIdx-1];
			   s_U[sj][si-1] =	  U[globalIdx-1];
			   s_V[sj][si-1] =    V[globalIdx-1];

			         s_h[sj][si + BLOCK_SIZE_x ] =    h[globalIdx + BLOCK_SIZE_x ];
				  s_upos[sj][si + BLOCK_SIZE_x ] = upos[globalIdx + BLOCK_SIZE_x ];
				  s_vpos[sj][si + BLOCK_SIZE_x ] = vpos[globalIdx + BLOCK_SIZE_x ];
				     s_U[sj][si + BLOCK_SIZE_x ] =    U[globalIdx + BLOCK_SIZE_x ];
				     s_V[sj][si + BLOCK_SIZE_x ] =    V[globalIdx + BLOCK_SIZE_x ];
		}
		if (threadIdx.y==0) {
			s_h[sj-1][si] =    h[globalIdx - n_d];
		 s_upos[sj-1][si] = upos[globalIdx - n_d];
		 s_vpos[sj-1][si] = vpos[globalIdx - n_d];
			s_U[sj-1][si] =    U[globalIdx - n_d];
			s_V[sj-1][si] =    V[globalIdx - n_d];

			s_h[sj+BLOCK_SIZE_y][si] =    h[globalIdx +n_d*(BLOCK_SIZE_y)];
		 s_upos[sj+BLOCK_SIZE_y][si] = upos[globalIdx +n_d*(BLOCK_SIZE_y)];
		 s_vpos[sj+BLOCK_SIZE_y][si] = vpos[globalIdx +n_d*(BLOCK_SIZE_y)];
			s_U[sj+BLOCK_SIZE_y][si] =    U[globalIdx +n_d*(BLOCK_SIZE_y)];
			s_V[sj+BLOCK_SIZE_y][si] =    V[globalIdx +n_d*(BLOCK_SIZE_y)];
		}

		// copy global variables into shared memory
		   s_h[sj][si] = h[globalIdx];
		s_upos[sj][si] = upos[globalIdx];
		s_vpos[sj][si] = vpos[globalIdx]; 
		   s_U[sj][si] = U[globalIdx];
		   s_V[sj][si] = V[globalIdx];

		__syncthreads();
		
		// fill in periodic images in shared memory array 
		//if (i < 4) {
		//	s_f[sj][si - 4] = s_f[sj][si + mx - 5];
		//	s_f[sj][si + mx] = s_f[sj][si + 1];
		//}

		//__syncthreads();

		//update Hx and Hy
	/*	 s_hx[sj][si] =
			s_upos[sj][si] * s_h[sj][si]
			+ (1 - s_upos[sj][si]) *s_h[sj][si + 1];

		s_hy[sj][si] =
			s_vpos[sj][si] * s_h[sj][si]
			+ (1 - s_vpos[sj][si]) *s_h[sj+1][si];*/

		//update U (no sync necessary)
		utemp = s_U[sj][si] - g * dt / dx * (s_h[sj][si + 1] - s_h[sj][si])
			      - s_upos[sj][si] * dt / dx * (s_U[sj][si] - s_U[sj][si - 1])*(s_U[sj][si] + s_U[sj][si - 1]) / 2
			      - s_vpos[sj][si] * dt / dy * (s_U[sj][si] - s_U[sj - 1][si])*(s_V[sj-1][si] + s_V[sj - 1][si+1]) / 2
			- (1 - s_upos[sj][si]) * dt / dx * (s_U[sj][si + 1] - s_U[sj][si])*(s_U[sj][si] + s_U[sj][si + 1]) / 2
			- (1 - s_vpos[sj][si]) * dt / dy * (s_U[sj + 1][si] - s_U[sj][si])*(s_V[sj][si] + s_V[sj ][si+1]) / 2;

		__syncthreads();

		//write temp values to shared memory after sync and update upos
		s_U[sj][si] = utemp;
		s_upos[sj][si] = (__int8)(utemp > 0);

		__syncthreads();
		//now that 
		 s_hx =
			          (s_upos[sj][si] * s_h[sj][si]
				+ (1 - s_upos[sj][si]) *s_h[sj][si + 1]);

		 s_hxmin =
			           (s_upos[sj][si - 1] * s_h[sj][si - 1]
				+ (1 - s_upos[sj][si - 1]) *s_h[sj][si]);

		//write back to global memory
		U[globalIdx] = utemp;

		//update V
		vtemp = s_V[sj][si] - g * dt / dy * (s_h[sj + 1][si] - s_h[sj][si])
			- s_vpos[sj][si] * dt / dy * (s_V[sj][si] - s_V[sj - 1][si])*(s_V[sj][si] + s_V[sj - 1][si]) / 2
			- s_upos[sj][si] * dt / dx * (s_V[sj][si] - s_V[sj][si-1])  *(s_U[sj + 1][si - 1] + s_U[sj][si - 1]) / 2
			- (1-s_vpos[sj][si]) * dt / dy * (s_V[sj+1][si] - s_V[sj ][si])*(s_V[sj][si] + s_V[sj + 1][si]) / 2
			- (1-s_upos[sj][si]) * dt / dx * (s_V[sj][si+1] - s_V[sj][si])  *(s_U[sj + 1][si ] + s_U[sj][si ]) / 2;

		__syncthreads();

		s_V[sj][si] =  vtemp;
		s_vpos[sj][si]= (__int8)(vtemp > 0);

		__syncthreads();

		V[globalIdx] = vtemp;
		
		//calculate hy
		s_hy =
			s_vpos[sj][si] * s_h[sj][si]
			+ (1 - s_vpos[sj][si]) *s_h[sj + 1][si];

		s_hymin =
			s_vpos[sj - 1][si] * s_h[sj - 1][si]
			+ (1 - s_vpos[sj - 1][si]) *s_h[sj][si];

		// update h
		s_h[sj][si] = s_h[sj][si] - dt / dx * (s_hx * s_U[sj][si] - s_hxmin * s_U[sj][si - 1])
								  - dt / dy * (s_hy * s_V[sj][si] - s_hymin * s_V[sj - 1][si]);

			/*s_h[sj][si] = s_h[sj][si] - dt / dx * s_hx[sj][si] * s_U[sj][si] - s_hx[sj][si - 1] * s_U[sj][si - 1]
				- dt / dy * s_hy[sj][si] * s_V[sj][si] - s_hy[sj - 1][si] * s_V[sj - 1][si];*/

		__syncthreads();
		//write h back to global memory
		h[globalIdx] = s_h[sj][si];
		
		__syncthreads();
			
	}

	__global__ void updatenobool(float *h, float *U, float *V, float dt)
	{


		__shared__      float s_h[BLOCK_SIZE_y + 2][BLOCK_SIZE_x + 2]; // 4-wide halo
		//__shared__     float s_hy[BLOCK_SIZE_y+2][BLOCK_SIZE_x+2]; // 4-wide halo
		//__shared__     float s_hx[BLOCK_SIZE_y+2][BLOCK_SIZE_x+2]; // 4-wide halo	
		__shared__      float s_U[BLOCK_SIZE_y + 2][BLOCK_SIZE_x + 2]; // 2-wide halo
		__shared__      float s_V[BLOCK_SIZE_y + 2][BLOCK_SIZE_x + 2]; // 2-wide halo

		//int i = threadIdx.x;
		//int j = blockIdx.x*blockDim.y + threadIdx.y;
		int j = blockIdx.y*blockDim.y + threadIdx.y;
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		int si = threadIdx.x + 1; // local i for shared memory access + halo offset
		int sj = threadIdx.y + 1; // local j for shared memory access
		float utemp, vtemp;
		float s_hx, s_hxmin, s_hy, s_hymin;
		__int8 s_upos, s_vpos;
		int globalIdx = (j + 1) * n_d + i + 1;

		//Boundaries
		if (threadIdx.x == 0) {

			s_h[sj][si - 1] = h[globalIdx - 1];
			s_U[sj][si - 1] = U[globalIdx - 1];
			s_V[sj][si - 1] = V[globalIdx - 1];

			s_h[sj][si + BLOCK_SIZE_x] = h[globalIdx + BLOCK_SIZE_x];
			s_U[sj][si + BLOCK_SIZE_x] = U[globalIdx + BLOCK_SIZE_x];
			s_V[sj][si + BLOCK_SIZE_x] = V[globalIdx + BLOCK_SIZE_x];
		}
		if (threadIdx.y == 0) {
			s_h[sj - 1][si] = h[globalIdx - n_d];
			
			s_U[sj - 1][si] = U[globalIdx - n_d];
			s_V[sj - 1][si] = V[globalIdx - n_d];

			s_h[sj + BLOCK_SIZE_y][si] = h[globalIdx + n_d * (BLOCK_SIZE_y)];
			
			s_U[sj + BLOCK_SIZE_y][si] = U[globalIdx + n_d * (BLOCK_SIZE_y)];
			s_V[sj + BLOCK_SIZE_y][si] = V[globalIdx + n_d * (BLOCK_SIZE_y)];
		}

		// copy global variables into shared memory
		s_h[sj][si] = h[globalIdx];
		
		s_U[sj][si] = U[globalIdx];
		s_V[sj][si] = V[globalIdx];
		s_upos = s_U[sj][si] >= 0;
		s_vpos = s_V[sj][si] >= 0;
		__syncthreads();
		
		
		// fill in periodic images in shared memory array 
		//if (i < 4) {
		//	s_f[sj][si - 4] = s_f[sj][si + mx - 5];
		//	s_f[sj][si + mx] = s_f[sj][si + 1];
		//}

		//__syncthreads();

		//update Hx and Hy
	/*	 s_hx[sj][si] =
			s_upos[sj][si] * s_h[sj][si]
			+ (1 - s_upos[sj][si]) *s_h[sj][si + 1];
		
		s_hy[sj][si] =
			s_vpos[sj][si] * s_h[sj][si]
			+ (1 - s_vpos[sj][si]) *s_h[sj+1][si];*/

			//update U (no sync necessary)
		utemp = s_U[sj][si] - g * dt / dx * (s_h[sj][si + 1] - s_h[sj][si])
			- s_upos * dt / dx * (s_U[sj][si] - s_U[sj][si - 1])*(s_U[sj][si] + s_U[sj][si - 1]) / 2
			- s_vpos * dt / dy * (s_U[sj][si] - s_U[sj - 1][si])*(s_V[sj - 1][si] + s_V[sj - 1][si + 1]) / 2
			- (1 - s_upos) * dt / dx * (s_U[sj][si + 1] - s_U[sj][si])*(s_U[sj][si] + s_U[sj][si + 1]) / 2
			- (1 - s_vpos) * dt / dy * (s_U[sj + 1][si] - s_U[sj][si])*(s_V[sj][si] + s_V[sj][si + 1]) / 2;

		__syncthreads();

		//write temp values to shared memory after sync and update upos
		s_U[sj][si] = utemp;
		s_upos = (utemp > 0);

		__syncthreads();
		//now that 
		s_hx =
			(s_upos * s_h[sj][si]
				+ (1 - s_upos) *s_h[sj][si + 1]);

		s_hxmin =
			((s_U[sj][si - 1]>=0) * s_h[sj][si - 1]
				+ (s_U[sj][si - 1]<0) *s_h[sj][si]);

		//write back to global memory
		U[globalIdx] = utemp;

		//update V
		vtemp = s_V[sj][si] - g * dt / dy * (s_h[sj + 1][si] - s_h[sj][si])
			- s_vpos * dt / dy * (s_V[sj][si] - s_V[sj - 1][si])*(s_V[sj][si] + s_V[sj - 1][si]) / 2
			- s_upos * dt / dx * (s_V[sj][si] - s_V[sj][si - 1])  *(s_U[sj + 1][si - 1] + s_U[sj][si - 1]) / 2
			- (1 - s_vpos) * dt / dy * (s_V[sj + 1][si] - s_V[sj][si])*(s_V[sj][si] + s_V[sj + 1][si]) / 2
			- (1 - s_upos) * dt / dx * (s_V[sj][si + 1] - s_V[sj][si])  *(s_U[sj + 1][si] + s_U[sj][si]) / 2;

		__syncthreads();

		s_V[sj][si] = vtemp;
		s_vpos = (vtemp >= 0);

		__syncthreads();

		V[globalIdx] = vtemp;

		//calculate hy
		s_hy =
			s_vpos * s_h[sj][si]
			+ (1 - s_vpos) *s_h[sj + 1][si];

		s_hymin =
			(s_V[sj - 1][si]>=0) * s_h[sj - 1][si]
			+ (s_V[sj - 1][si] < 0) *s_h[sj][si];

		__syncthreads();
		// update h
		s_h[sj][si] = s_h[sj][si] - dt / dx * (s_hx * s_U[sj][si] - s_hxmin * s_U[sj][si - 1])
			- dt / dy * (s_hy * s_V[sj][si] - s_hymin * s_V[sj - 1][si]);

		/*s_h[sj][si] = s_h[sj][si] - dt / dx * s_hx[sj][si] * s_U[sj][si] - s_hx[sj][si - 1] * s_U[sj][si - 1]
			- dt / dy * s_hy[sj][si] * s_V[sj][si] - s_hy[sj - 1][si] * s_V[sj - 1][si];*/

		__syncthreads();
		//write h back to global memory
		h[globalIdx] = s_h[sj][si];

		__syncthreads();

	}

	__global__ void updateborders(float *h, float *U, float *V, float dt, int iter)
	{


		__shared__      float s_h[BLOCK_SIZE_y + 2][BLOCK_SIZE_x + 2]; // 4-wide halo
		//__shared__     float s_hy[BLOCK_SIZE_y+2][BLOCK_SIZE_x+2]; // 4-wide halo
		//__shared__     float s_hx[BLOCK_SIZE_y+2][BLOCK_SIZE_x+2]; // 4-wide halo	
		__shared__      float s_U[BLOCK_SIZE_y + 2][BLOCK_SIZE_x + 2]; // 2-wide halo
		__shared__      float s_V[BLOCK_SIZE_y + 2][BLOCK_SIZE_x + 2]; // 2-wide halo

		//int i = threadIdx.x;
		//int j = blockIdx.x*blockDim.y + threadIdx.y;
		int j = blockIdx.y*blockDim.y + threadIdx.y;
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		int si = threadIdx.x + 1; // local i for shared memory access + halo offset
		int sj = threadIdx.y + 1; // local j for shared memory access
		float utemp, vtemp;
		float s_hx, s_hxmin, s_hy, s_hymin;
		__int8 s_upos, s_vpos;
		int globalIdx = (j + 1) * n_d + i + 1;

		//Boundaries
		if (threadIdx.x == 0) {

			s_h[sj][si - 1] = h[globalIdx - 1];
			s_U[sj][si - 1] = U[globalIdx - 1];
			s_V[sj][si - 1] = V[globalIdx - 1];

			s_h[sj][si + BLOCK_SIZE_x] = h[globalIdx + BLOCK_SIZE_x];
			s_U[sj][si + BLOCK_SIZE_x] = U[globalIdx + BLOCK_SIZE_x];
			s_V[sj][si + BLOCK_SIZE_x] = V[globalIdx + BLOCK_SIZE_x];
		}
		if (threadIdx.y == 0) {
			s_h[sj - 1][si] = h[globalIdx - n_d];
			s_U[sj - 1][si] = U[globalIdx - n_d];
			s_V[sj - 1][si] = V[globalIdx - n_d];

			s_h[sj + BLOCK_SIZE_y][si] = h[globalIdx + n_d * (BLOCK_SIZE_y)];
			s_U[sj + BLOCK_SIZE_y][si] = U[globalIdx + n_d * (BLOCK_SIZE_y)];
			s_V[sj + BLOCK_SIZE_y][si] = V[globalIdx + n_d * (BLOCK_SIZE_y)];
		}

		// copy global variables into shared memory
		s_h[sj][si] = h[globalIdx];
		s_U[sj][si] = U[globalIdx];
		s_V[sj][si] = V[globalIdx];
		s_upos = s_U[sj][si] >= 0;
		s_vpos = s_V[sj][si] >= 0;

		__syncthreads();

		for (int k = 0; k < iter; k++) {

			if (threadIdx.x == 0) {

				s_h[sj][si - 1] = h[globalIdx - 1];				
				s_U[sj][si - 1] = U[globalIdx - 1];
				s_V[sj][si - 1] = V[globalIdx - 1];
				s_h[sj][si + BLOCK_SIZE_x] = h[globalIdx + BLOCK_SIZE_x];
				s_U[sj][si + BLOCK_SIZE_x] = U[globalIdx + BLOCK_SIZE_x];
				s_V[sj][si + BLOCK_SIZE_x] = V[globalIdx + BLOCK_SIZE_x];
			}
			if (threadIdx.y == 0) {
				s_h[sj - 1][si] = h[globalIdx - n_d];				
				s_U[sj - 1][si] = U[globalIdx - n_d];
				s_V[sj - 1][si] = V[globalIdx - n_d];
				s_h[sj + BLOCK_SIZE_y][si] = h[globalIdx + n_d * (BLOCK_SIZE_y)];				
				s_U[sj + BLOCK_SIZE_y][si] = U[globalIdx + n_d * (BLOCK_SIZE_y)];
				s_V[sj + BLOCK_SIZE_y][si] = V[globalIdx + n_d * (BLOCK_SIZE_y)];
			}
			

				//update U (no sync necessary)
			utemp = s_U[sj][si] - g * dt / dx * (s_h[sj][si + 1] - s_h[sj][si])
				- s_upos * dt / dx * (s_U[sj][si] - s_U[sj][si - 1])*(s_U[sj][si] + s_U[sj][si - 1]) / 2
				- s_vpos * dt / dy * (s_U[sj][si] - s_U[sj - 1][si])*(s_V[sj - 1][si] + s_V[sj - 1][si + 1]) / 2
				- (1 - s_upos) * dt / dx * (s_U[sj][si + 1] - s_U[sj][si])*(s_U[sj][si] + s_U[sj][si + 1]) / 2
				- (1 - s_vpos) * dt / dy * (s_U[sj + 1][si] - s_U[sj][si])*(s_V[sj][si] + s_V[sj][si + 1]) / 2;

			__syncthreads();

			//write temp values to shared memory after sync and update upos
			s_U[sj][si] = utemp;
			s_upos = (utemp > 0);

			__syncthreads();
			//now that 
			s_hx =
				(s_upos * s_h[sj][si]
					+ (1 - s_upos) *s_h[sj][si + 1]);

			s_hxmin =
				((s_U[sj][si - 1] >= 0) * s_h[sj][si - 1]
					+ (s_U[sj][si - 1] < 0) *s_h[sj][si]);

			//write back to global memory
		//	U[globalIdx] = utemp;

			//update V
			vtemp = s_V[sj][si] - g * dt / dy * (s_h[sj + 1][si] - s_h[sj][si])
				- s_vpos * dt / dy * (s_V[sj][si] - s_V[sj - 1][si])*(s_V[sj][si] + s_V[sj - 1][si]) / 2
				- s_upos * dt / dx * (s_V[sj][si] - s_V[sj][si - 1])  *(s_U[sj + 1][si - 1] + s_U[sj][si - 1]) / 2
				- (1 - s_vpos) * dt / dy * (s_V[sj + 1][si] - s_V[sj][si])*(s_V[sj][si] + s_V[sj + 1][si]) / 2
				- (1 - s_upos) * dt / dx * (s_V[sj][si + 1] - s_V[sj][si])  *(s_U[sj + 1][si] + s_U[sj][si]) / 2;

			

			__syncthreads();

			s_vpos = (vtemp >= 0);
			s_V[sj][si] = vtemp;
			

			__syncthreads();

			//V[globalIdx] = vtemp;

			//calculate hy
			s_hy =
				s_vpos * s_h[sj][si]
				+ (1 - s_vpos) *s_h[sj + 1][si];

			s_hymin =
				(s_V[sj - 1][si] >= 0) * s_h[sj - 1][si]
				+ (s_V[sj - 1][si] < 0) *s_h[sj][si];

			__syncthreads();
			// update h
			s_h[sj][si] = s_h[sj][si] - dt / dx * (s_hx * s_U[sj][si] - s_hxmin * s_U[sj][si - 1])
				- dt / dy * (s_hy * s_V[sj][si] - s_hymin * s_V[sj - 1][si]);

			/*s_h[sj][si] = s_h[sj][si] - dt / dx * s_hx[sj][si] * s_U[sj][si] - s_hx[sj][si - 1] * s_U[sj][si - 1]
				- dt / dy * s_hy[sj][si] * s_V[sj][si] - s_hy[sj - 1][si] * s_V[sj - 1][si];*/

			__syncthreads();
			
			if (threadIdx.x == 0) {

				h[globalIdx] = s_h[sj][si];				
				U[globalIdx] =  utemp;
				V[globalIdx] = vtemp;

				h[globalIdx + BLOCK_SIZE_x-1] = s_h[sj][si + BLOCK_SIZE_x-1];
				U[globalIdx + BLOCK_SIZE_x - 1] =  s_U[sj][si + BLOCK_SIZE_x - 1];
				V[globalIdx + BLOCK_SIZE_x-1] = s_V[sj][si + BLOCK_SIZE_x-1];
			}
			if (threadIdx.y == 0) {
				h[globalIdx ] = s_h[sj ][si];
				U[globalIdx] =  s_U[sj][si];
				V[globalIdx ] = s_V[sj ][si];

				h[globalIdx + (n_d) * (BLOCK_SIZE_y-1)] = s_h[sj + BLOCK_SIZE_y-1][si];
				U[globalIdx + (n_d) * (BLOCK_SIZE_y - 1)] =  s_U[sj + BLOCK_SIZE_y - 1][si];
				V[globalIdx + (n_d) * (BLOCK_SIZE_y-1)] = s_V[sj + BLOCK_SIZE_y-1][si];
			}
			__syncthreads();
			
		}
		V[globalIdx] = vtemp;
		//U[globalIdx] = utemp;
		h[globalIdx] = s_h[sj][si];
	}

	/* void syncthreads(int threadcount)
	{
		std::condition_variable wake;
		
		std::unique_lock<std::mutex> lck(mu);

		barrier++;		

		
			

		if (barrier == threadcount)
		{
			barrier = 0;			
			wake.notify_all();
			
			
			
		}
		else
			while(barrier<threadcount)
		wake.wait(lck);
		
		assert(barrier != threadcount);

		if (barrier == --threadcount)
		{
			wake.notify_all();
		}
		else
		{
			wake.wait(lck, [this]() { return barrier == threadcount; });
		}



	} 
	*/

	void updatecputhread(float* h, float* U, float* V, float dt, int tid, int numthreads,int iter, cbar::cyclicbarrier* cb)
	{
		int BLOCK_SIZE_x = n-2;
		int BLOCK_SIZE_y = ceil((float)(n-2) / (float)numthreads);
		
		float g = g_h;
		float dx = dx_h;
		float dy = dy_h;

		float** s_h = NULL; float** s_U = NULL; float** s_V = NULL; __int8** s_upos = NULL; __int8** s_vpos = NULL;
		s_h = new float* [(BLOCK_SIZE_y + 2)]; s_U = new float* [(BLOCK_SIZE_y + 2)]; s_V = new float* [(BLOCK_SIZE_y + 2)];
		s_upos = new __int8* [(BLOCK_SIZE_y + 2)]; s_vpos = new __int8* [(BLOCK_SIZE_y + 2)];

				// Create a row for every pointer 
				for (int k = 0; k <= BLOCK_SIZE_y+2; k++)
		{ 
			   s_h[k] = new float[BLOCK_SIZE_x+2];
			   s_U[k] = new float[BLOCK_SIZE_x+2];
			   s_V[k] = new float[BLOCK_SIZE_x+2];
			s_upos[k] = new __int8[BLOCK_SIZE_x + 2];
			s_vpos[k] = new __int8[BLOCK_SIZE_x + 2];
		}

				for (int z = 0; z < iter; z++) {

					for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
						for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
							int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;

							//Boundaries
							if (si == 1) {

								s_h[sj][si - 1] = h[globalIdx - 1];
								s_U[sj][si - 1] = U[globalIdx - 1];
								s_V[sj][si - 1] = V[globalIdx - 1];

								s_h[sj][si + BLOCK_SIZE_x] = h[globalIdx + BLOCK_SIZE_x];
								s_U[sj][si + BLOCK_SIZE_x] = U[globalIdx + BLOCK_SIZE_x];
								s_V[sj][si + BLOCK_SIZE_x] = V[globalIdx + BLOCK_SIZE_x];
							}
							if (sj == 1) {
								s_h[sj - 1][si] = h[globalIdx - n];
								s_U[sj - 1][si] = U[globalIdx - n];
								s_V[sj - 1][si] = V[globalIdx - n];

								s_h[sj + BLOCK_SIZE_y][si] = h[globalIdx + n * (BLOCK_SIZE_y)];
								s_U[sj + BLOCK_SIZE_y][si] = U[globalIdx + n * (BLOCK_SIZE_y)];
								s_V[sj + BLOCK_SIZE_y][si] = V[globalIdx + n * (BLOCK_SIZE_y)];
							}

							// copy global variables into shared memory
							s_h[sj][si] = h[globalIdx];
							s_U[sj][si] = U[globalIdx];
							s_V[sj][si] = V[globalIdx];
							s_upos[sj][si] = s_U[sj][si] >= 0;
							s_vpos[sj][si] = s_V[sj][si] >= 0;
						}
					}

					cb->await();//

						//update U
					for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
						for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
							int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;

							//update U (no sync necessary)
							U[globalIdx] = s_U[sj][si] - g * dt / dx * (s_h[sj][si + 1] - s_h[sj][si])
								- s_upos[sj][si] * dt / dx * (s_U[sj][si] - s_U[sj][si - 1]) * (s_U[sj][si] + s_U[sj][si - 1]) / 2
								- s_vpos[sj][si] * dt / dy * (s_U[sj][si] - s_U[sj - 1][si]) * (s_V[sj - 1][si] + s_V[sj - 1][si + 1]) / 2
								- (1 - s_upos[sj][si]) * dt / dx * (s_U[sj][si + 1] - s_U[sj][si]) * (s_U[sj][si] + s_U[sj][si + 1]) / 2
								- (1 - s_vpos[sj][si]) * dt / dy * (s_U[sj + 1][si] - s_U[sj][si]) * (s_V[sj][si] + s_V[sj][si + 1]) / 2;

							s_upos[sj][si] = (U[globalIdx] >= 0);
						}
					}
					cb->await();//syncthreads(numthreads);

					//write temp values to shared memory after sync and update upos
					for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
						for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
							int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;
							s_U[sj][si] = U[globalIdx];
							
						}
					}
					cb->await();//syncthreads(numthreads);

					//update V
					for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
						for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
							int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;
							V[globalIdx] = s_V[sj][si] - g * dt / dy * (s_h[sj + 1][si] - s_h[sj][si])
								- s_vpos[sj][si] * dt / dy * (s_V[sj][si] - s_V[sj - 1][si]) * (s_V[sj][si] + s_V[sj - 1][si]) / 2
								- s_upos[sj][si] * dt / dx * (s_V[sj][si] - s_V[sj][si - 1]) * (s_U[sj + 1][si - 1] + s_U[sj][si - 1]) / 2
								- (1 - s_vpos[sj][si]) * dt / dy * (s_V[sj + 1][si] - s_V[sj][si]) * (s_V[sj][si] + s_V[sj + 1][si]) / 2
								- (1 - s_upos[sj][si]) * dt / dx * (s_V[sj][si + 1] - s_V[sj][si]) * (s_U[sj + 1][si] + s_U[sj][si]) / 2;

							s_vpos[sj][si] = (V[globalIdx] >= 0);
						}
					}
					cb->await();//syncthreads(numthreads);


					for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
						for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
							int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;
							s_V[sj][si] = V[globalIdx];
							
						}
					}
					cb->await();//syncthreads(numthreads);
					//update H
					for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
						for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
							int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;
							//calculate hy
							float s_hy =
								s_vpos[sj][si] * s_h[sj][si]
								+ (1 - s_vpos[sj][si]) * s_h[sj + 1][si];

							float s_hymin =
								(s_V[sj - 1][si] >= 0) * s_h[sj - 1][si]
								+ (s_V[sj - 1][si] < 0) * s_h[sj][si];

							float s_hx =
								(s_upos[sj][si] * s_h[sj][si]
									+ (1 - s_upos[sj][si]) * s_h[sj][si + 1]);

							float s_hxmin =
								((s_U[sj][si - 1] >= 0) * s_h[sj][si - 1]
									+ (s_U[sj][si - 1] < 0) * s_h[sj][si]);


							// update h
							h[globalIdx] = s_h[sj][si] - dt / dx * (s_hx * s_U[sj][si] - s_hxmin * s_U[sj][si - 1])
								- dt / dy * (s_hy * s_V[sj][si] - s_hymin * s_V[sj - 1][si]);




						}

					}
					cb->await();//syncthreads(numthreads);
				}
	}

	void updatecputhreadborder(float* h, float* U, float* V, float dt, int tid, int numthreads, int iter, cbar::cyclicbarrier* cb)
	{  
		const int BLOCK_SIZE_x = n - 2;
		const int BLOCK_SIZE_y = ceil((float)(n - 2) / (float)numthreads);

		const float g = g_h;
		const float dx = dx_h;
		const float dy = dy_h;

		float** s_h = NULL; float** s_U = NULL; float** s_V = NULL; __int8** s_upos = NULL; __int8** s_vpos = NULL;
		s_h = new float*[(BLOCK_SIZE_y + 2)]; s_U = new float*[(BLOCK_SIZE_y + 2)]; s_V = new float*[(BLOCK_SIZE_y + 2)];
		s_upos = new __int8*[(BLOCK_SIZE_y + 2)]; s_vpos = new __int8*[(BLOCK_SIZE_y + 2)];

		// Create a row for every pointer 
		for (int k = 0; k <= BLOCK_SIZE_y + 2; k++)
		{
			s_h[k] = new float[BLOCK_SIZE_x + 2];
			s_U[k] = new float[BLOCK_SIZE_x + 2];
			s_V[k] = new float[BLOCK_SIZE_x + 2];
			s_upos[k] = new __int8[BLOCK_SIZE_x + 2];
			s_vpos[k] = new __int8[BLOCK_SIZE_x + 2];
		}

        
		
			for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
				for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
				int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;
				// copy global variables into shared memory
				s_h[sj][si] = h[globalIdx];
				s_U[sj][si] = U[globalIdx];
				s_V[sj][si] = V[globalIdx];
				s_upos[sj][si] = s_U[sj][si] >= 0;
				s_vpos[sj][si] = s_V[sj][si] >= 0;
			}
		}
		for (int z = 0; z < iter; z++) {
			
			
				for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
					for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
					int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;

					//Boundaries
					if (si == 1) {

						s_h[sj][si - 1] = h[globalIdx - 1];
						s_U[sj][si - 1] = U[globalIdx - 1];
						s_V[sj][si - 1] = V[globalIdx - 1];

						s_h[sj][si + BLOCK_SIZE_x] = h[globalIdx + BLOCK_SIZE_x];
						s_U[sj][si + BLOCK_SIZE_x] = U[globalIdx + BLOCK_SIZE_x];
						s_V[sj][si + BLOCK_SIZE_x] = V[globalIdx + BLOCK_SIZE_x];
					}
					if (sj == 1) {
						s_h[sj - 1][si] = h[globalIdx - n];
						s_U[sj - 1][si] = U[globalIdx - n];
						s_V[sj - 1][si] = V[globalIdx - n];

						s_h[sj + BLOCK_SIZE_y][si] = h[globalIdx + n * (BLOCK_SIZE_y)];
						s_U[sj + BLOCK_SIZE_y][si] = U[globalIdx + n * (BLOCK_SIZE_y)];
						s_V[sj + BLOCK_SIZE_y][si] = V[globalIdx + n * (BLOCK_SIZE_y)];
					}

					
				}
			}

			cb->await();//

				//update U

			for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
			for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
				
					int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;

					//update U (no sync necessary)
					U[globalIdx] = s_U[sj][si] - g * dt / dx * (s_h[sj][si + 1] - s_h[sj][si])
						- s_upos[sj][si] * dt / dx * (s_U[sj][si] - s_U[sj][si - 1]) * (s_U[sj][si] + s_U[sj][si - 1]) / 2
						- s_vpos[sj][si] * dt / dy * (s_U[sj][si] - s_U[sj - 1][si]) * (s_V[sj - 1][si] + s_V[sj - 1][si + 1]) / 2
						- (1 - s_upos[sj][si]) * dt / dx * (s_U[sj][si + 1] - s_U[sj][si]) * (s_U[sj][si] + s_U[sj][si + 1]) / 2
						- (1 - s_vpos[sj][si]) * dt / dy * (s_U[sj + 1][si] - s_U[sj][si]) * (s_V[sj][si] + s_V[sj][si + 1]) / 2;

					s_upos[sj][si] = (U[globalIdx] >= 0);
				}
			}
			cb->await();//syncthreads(numthreads);

			//write temp values to shared memory after sync and update upos
			for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
			for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
				
					int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;
					s_U[sj][si] = U[globalIdx];

				}
			}
			cb->await();//syncthreads(numthreads);

			//update V
			
				for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
					for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
					int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;
					V[globalIdx] = s_V[sj][si] - g * dt / dy * (s_h[sj + 1][si] - s_h[sj][si])
						- s_vpos[sj][si] * dt / dy * (s_V[sj][si] - s_V[sj - 1][si]) * (s_V[sj][si] + s_V[sj - 1][si]) / 2
						- s_upos[sj][si] * dt / dx * (s_V[sj][si] - s_V[sj][si - 1]) * (s_U[sj + 1][si - 1] + s_U[sj][si - 1]) / 2
						- (1 - s_vpos[sj][si]) * dt / dy * (s_V[sj + 1][si] - s_V[sj][si]) * (s_V[sj][si] + s_V[sj + 1][si]) / 2
						- (1 - s_upos[sj][si]) * dt / dx * (s_V[sj][si + 1] - s_V[sj][si]) * (s_U[sj + 1][si] + s_U[sj][si]) / 2;

					s_vpos[sj][si] = (V[globalIdx] >= 0);
				}
			}
			cb->await();//syncthreads(numthreads);


			for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
			for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
				
					int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;
					s_V[sj][si] = V[globalIdx];

				}
			}
			cb->await();//syncthreads(numthreads);
			//update H
			for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
			for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
				
					int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;
					//calculate hy
					float s_hy =
						s_vpos[sj][si] * s_h[sj][si]
						+ (1 - s_vpos[sj][si]) * s_h[sj + 1][si];

					float s_hymin =
						(s_V[sj - 1][si] >= 0) * s_h[sj - 1][si]
						+ (s_V[sj - 1][si] < 0) * s_h[sj][si];

					float s_hx =
						(s_upos[sj][si] * s_h[sj][si]
							+ (1 - s_upos[sj][si]) * s_h[sj][si + 1]);

					float s_hxmin =
						((s_U[sj][si - 1] >= 0) * s_h[sj][si - 1]
							+ (s_U[sj][si - 1] < 0) * s_h[sj][si]);


					// update h
					h[globalIdx] = s_h[sj][si] - dt / dx * (s_hx * s_U[sj][si] - s_hxmin * s_U[sj][si - 1])
						- dt / dy * (s_hy * s_V[sj][si] - s_hymin * s_V[sj - 1][si]);
					



				}

			}
			cb->await();//syncthreads(numthreads);
			
				for (int sj = 1; sj < BLOCK_SIZE_y + 1; sj++) {
					for (int si = 1; si < BLOCK_SIZE_x + 1; si++) {
					int globalIdx = si + (sj + BLOCK_SIZE_y * tid) *n;
					s_h[sj][si] = h[globalIdx];

				}
			}
			cb->await();
		}
	}
	
	__int8 *initializeBoolArray() {
		__int8 *ptr = 0;
		//printf("Initializing bool array \n");
		checkCuda(hipMalloc(&ptr, n * n * sizeof(__int8)));
		//checkCudaError("Malloc for matrix on device failed !");

		return ptr;

	}

	float *initializeFloatArray(){
 //   void initializearrays(float *H, float  *U, float  *V, /*float  *Hx, float  *Hy,*/ __int8 *Upos, __int8 *Vpos) {
		float *ptr = 0;
		//printf("Initializing float array \n");
	checkCuda(hipMalloc(&ptr, n * n * sizeof(float)));
	//checkCudaError("Malloc for matrix on device failed !");

	return ptr;
	

	}

	__global__	void fillarrays(float *H, /*float  *Hx, float  *Hy,*/ __int8 *Upos, __int8 *Vpos) {

		//int i = threadIdx.x;
		//int j = blockIdx.x*blockDim.y + threadIdx.y;
		int j = blockIdx.y*blockDim.y + threadIdx.y;
		int i = blockIdx.x*blockDim.x + threadIdx.x;		
		int globalIdx = (j+1) * n_d + i+1;
		//printf("globalidx = %d \n", globalIdx);
		       H[globalIdx] = Hstart;
			  // U[globalIdx] = 0;
			   //V[globalIdx] = 0;
			Upos[globalIdx] = 1;
			Vpos[globalIdx] = 1;
			if (i == 0) {
			    H[globalIdx-1] = Hstart;			
		     Upos[globalIdx-1] = 1;
			 Vpos[globalIdx-1] = 1;
			    H[globalIdx +n_d-2] = Hstart;
			 Upos[globalIdx +n_d-2] = 1;
			 Vpos[globalIdx +n_d-2] = 1;
			}
			if (j == 0) {
				   H[globalIdx - n_d] = Hstart;
				Upos[globalIdx - n_d] = 1;
				Vpos[globalIdx - n_d] = 1;
				   H[globalIdx + n_d*(n_d-2) ] = Hstart;
				Upos[globalIdx + n_d*(n_d-2) ] = 1;
				Vpos[globalIdx + n_d*(n_d-2) ] = 1;
			}
			__syncthreads();

	}

	__global__	void printIdx() {

		//int i = threadIdx.x;
		//int j = blockIdx.x*blockDim.y + threadIdx.y;
		int j = blockIdx.y*blockDim.y + threadIdx.y;
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		int globalIdx = j * n_d + i;
		//printf("globalidx = %d \n", globalIdx);
		printf("i = %d \n j= %d \n globalidx = %d \n", i,j,globalIdx);

		

	}

	__global__ void Waterdrop(float *H,float height, int width, float step) {

		
		int j = blockIdx.y*blockDim.y + threadIdx.y;
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		float x = -1 + i * step;
		float y = -1 + j * step;
		
		
		float D = (1+4*droppar)/5* height *expf(-5*(x*x + y * y));
		int globalIdx = (j + 1 + droppar * (n_d - width)) * n_d + i + 1 + droppar * (n_d - width);
		 
		H[globalIdx] = H[globalIdx] + D;
		//printf("globalIdx: %d x: %.3f y: %.3f ", globalIdx, x, y);
		//printf("Idx: %d x: %d y: %d D: %d ", globalIdx, x , y ,  D);
		//printf("x: %.2f y: %.2f D: %.2f \n ",  x , y ,  D);
		//printf("i: %d j: %d \n" ,i ,j);
		//printf("D: %.3f \n" ,D);
		__syncthreads();
		//float height = 1.5*Hstart;
		//int width = (int)(n-2)/2;
		
		//for (float i = -1; i < 1; i = i + 2 / (width - 1));
		/*[x, y] = ndgrid(-1:(2 / (width - 1)) : 1);
		D = height * exp(-5 * (x. ^ 2 + y. ^ 2));
		w = size(D, 1);
		i = ceil(rand*(n - w)) + (1:w);
		j = ceil(rand*(n - w)) + (1:w);
		H(i, j) = H(i, j) + (1 + 4 * rand) / 5 * D;*/
   }

void copyConstants(){

    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(n_d), &n, sizeof(int), 0, hipMemcpyHostToDevice)); //grid size
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(g), &g_h, sizeof(float), 0, hipMemcpyHostToDevice));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(dx), &dx_h, sizeof(float), 0, hipMemcpyHostToDevice));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(dy), &dy_h, sizeof(float), 0, hipMemcpyHostToDevice));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(cf), &cf_h, sizeof(float), 0, hipMemcpyHostToDevice));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(Hstart), &Hstart_h, sizeof(float), 0, hipMemcpyHostToDevice));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(droppar), &droppar_h, sizeof(float), 0, hipMemcpyHostToDevice));
}

void CudaCheckError() {
	// make the host block until the device is finished with foo
	hipDeviceSynchronize();

	// check for error
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	}

void initializeWaterdrop(float *H) {

	int width = (int)floor(((float)n - 2) / 3);
	dim3 gridSizeDrop(ceil((float)width / 32), ceil((float)width / 32));
	dim3 blockSizeDrop(fmin(32,width),fmin(32,width));
	float height = 1.5*Hstart_h;
	float step = 2 / (float(width)-1);
	
	Waterdrop << <gridSizeDrop, blockSizeDrop >> > (H, height, width, step);
	
	CudaCheckError();
	
}

void showMatrix(const char *name, float *a, int n, int m)
{

	long x, y;

	for (y = 0; y < m; y++)
	{
		for (x = 0; x < n; x++){
			//printf("%s[%02ld][%02ld]=%6.2f  ", name, y, x, a[y*n + x]);
			if (a[y*n + x] == 0 || a[y*n + x] == 1)
			{
				printf("%d  ", (int)a[y*n + x]);
			}
			else {
				printf("%6.2f  ", a[y*n + x]);
			}
			}
		printf("\n");
	}
	printf("\n");

}

void copyfromCudaMatrix(float *h_a, float *d_a, int n, int m)
{
	//printf("Copying result back... ");
	checkCuda(hipMemcpy(h_a, d_a, n * m * sizeof(float), hipMemcpyDeviceToHost));
	//printf("success! \n");
	//checkCudaError("Matrix copy from device failed !");
}

void runprogram(int iter)
{

	
	// Print device and precision
	hipDeviceProp_t prop;
	checkCuda(hipGetDeviceProperties(&prop, 0));
	printf("\nDevice Name: %s\n", prop.name);
	printf("Compute Capability: %d.%d\n\n", prop.major, prop.minor);	
	printf("Shared Memory per Block: %d bytes \n", prop.sharedMemPerBlock);
	printf("Shared Memory per SM: %d bytes \n", prop.sharedMemPerMultiprocessor);
	printf("SM count: %d \n", prop.multiProcessorCount);
	printf("Max threads per SM: %d \n", prop.maxThreadsPerMultiProcessor);
	printf("Max threads per block: %d \n", prop.maxThreadsPerBlock);
	printf("block size: %d \n", BLOCK_SIZE_x*BLOCK_SIZE_y);
	printf("Max registers per thread: %d \n", prop.regsPerMultiprocessor / (prop.maxThreadsPerMultiProcessor / (BLOCK_SIZE_x*BLOCK_SIZE_y)) / (BLOCK_SIZE_x*BLOCK_SIZE_y));
	dim3 gridSize((n-2) / (BLOCK_SIZE_x), (n-2) / BLOCK_SIZE_y);
	dim3 blockSize(BLOCK_SIZE_x, BLOCK_SIZE_y);
	int blockmem = ((BLOCK_SIZE_y + 2)*(BLOCK_SIZE_x + 2)*(3 * sizeof(float) + 2 * sizeof(__int8)));
	printf("Block memory: %d bytes \n", blockmem);
	//check if blocks fit in shared memory:
	if (blockmem > prop.sharedMemPerBlock) {
		throw "Block size too large!! \n";
	}
	
	if ((prop.maxThreadsPerMultiProcessor / (BLOCK_SIZE_x*BLOCK_SIZE_y))*blockmem < 16* pow(2, 10)  )
	{
		checkCuda(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		printf("Configured for 48kb of L1 cache \n");
		
	}
	printf("n = %d \n", n);
	if (n * n * (3 * sizeof(float) + 2 * sizeof(__int8)) > prop.totalGlobalMem)
	{
		throw "Device out of memory!! max size = %d " ,  sqrt( prop.totalGlobalMem/ (3 * sizeof(float) + 2 * sizeof(__int8) ) );
	}
	float *H = initializeFloatArray();
	float *U = initializeFloatArray();
	float *V = initializeFloatArray();
	__int8 *Upos = initializeBoolArray();
	__int8 *Vpos = initializeBoolArray();
	
	copyConstants();

	printf("filling arrays... ");
		fillarrays << <gridSize, blockSize >> > (H,  Upos, Vpos);
		CudaCheckError();
		printf("success! \n");

		float *H_h = 0;
		hipHostMalloc(&H_h, n * n * sizeof(float));

		printf("initializing water drop... ");
		initializeWaterdrop(H);
		printf("success! \n");

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start);

		for (int i = 0; i < iter; i++) {
			update << <gridSize, blockSize >> > (H, Upos, Vpos, U, V, dt);
			//hipDeviceSynchronize();
		}

		//hipDeviceSynchronize();
		CudaCheckError();

		
	
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
	
		printf("update time: %.7f \n" , milliseconds/1000);
		copyfromCudaMatrix(H_h, H, n, n);
		if (n == 34)
		showMatrix("H", H_h, n, n);

		checkCuda(hipFree(H));
		checkCuda(hipFree(U));
		checkCuda(hipFree(V));
		checkCuda(hipFree(Upos));
		checkCuda(hipFree(Vpos));
		hipDeviceReset();
}

void runprogrambenchmark( int iter, int updatetype)
{

	hipEvent_t start2;
	hipEventCreate(&start2);
	hipEventRecord(start2);

	// Print device and precision
	hipDeviceProp_t prop;
	checkCuda(hipGetDeviceProperties(&prop, 0));

	dim3 gridSize((n - 2) / (BLOCK_SIZE_x), (n - 2) / BLOCK_SIZE_y);
	dim3 blockSize(BLOCK_SIZE_x, BLOCK_SIZE_y);
	int blockmem = ((BLOCK_SIZE_y + 2)*(BLOCK_SIZE_x + 2)*(3 * sizeof(float) + 2 * sizeof(__int8)));
	//check if blocks fit in shared memory:
	if (blockmem > prop.sharedMemPerBlock) {
		throw "Block size too large!! \n";
	}

	if ((prop.maxThreadsPerMultiProcessor / (BLOCK_SIZE_x*BLOCK_SIZE_y))*blockmem < 16 * pow(2, 10))
	{
		checkCuda(hipDeviceSetCacheConfig(hipFuncCachePreferL1));


	}

	if (n * n * (3 * sizeof(float) + 2 * sizeof(__int8)) > prop.totalGlobalMem)
	{
		throw "Device out of memory!! max size = %d ", sqrt(prop.totalGlobalMem / (3 * sizeof(float) + 2 * sizeof(__int8)));
	}
	float *H = initializeFloatArray();
	float *U = initializeFloatArray();
	float *V = initializeFloatArray();
	__int8 *Upos = initializeBoolArray();
	__int8 *Vpos = initializeBoolArray();

	copyConstants();


	fillarrays << <gridSize, blockSize >> > (H, Upos, Vpos);
	CudaCheckError();


	float *H_h = 0;
	hipHostMalloc(&H_h, n * n * sizeof(float));
	initializeWaterdrop(H);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	if (updatetype == 1) {
		for (int i = 0; i < iter; i++) {
			update << <gridSize, blockSize >> > (H, Upos, Vpos, U, V, dt);

		}
	}
	else if (updatetype == 2){
		for (int i = 0; i < iter; i++) {
			updatenobool << <gridSize, blockSize >> > (H, U, V, dt);

		}
		}
	else if (updatetype ==3)
		updateborders << <gridSize, blockSize >> > (H, U, V, dt, iter);
	
	/*switch (updatetype) {
	case 1:
		hipEventRecord(start);
		
		hipEventRecord(stop);
		hipEventSynchronize(stop);
	case 2:
		hipEventRecord(start);
		for (int i = 0; i < iter; i++) {
			updatenobool << <gridSize, blockSize >> > (H, U, V, dt);
			hipDeviceSynchronize();
		}
		hipEventRecord(stop);
		hipEventSynchronize(stop);
	case 3:
		hipEventRecord(start);
		updateborders << <gridSize, blockSize >> > (H, U, V, dt, iter);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
	}*/
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	CudaCheckError();
	if (n == 34) {
		copyfromCudaMatrix(H_h, H, n, n);
		showMatrix("H", H_h, n, n);
	}


	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("%.9f ", milliseconds/1000 );

	
	
	float milliseconds2 = 0;
	hipEventElapsedTime(&milliseconds2, start2, stop);
	printf("%.9f ", milliseconds2 / 1000);

	hipEventDestroy(start);
	hipEventDestroy(start2);
	hipEventDestroy(stop);
	checkCuda(hipFree(H));
	checkCuda(hipFree(U));
	checkCuda(hipFree(V));
	checkCuda(hipFree(Upos));
	checkCuda(hipFree(Vpos));
	hipDeviceReset();
}

void RunProgramCPU(int iter,int maxthreads) {

	//std::mutex mu;
	//int barrier = 0;
	// Print device and precision
	hipDeviceProp_t prop;
	checkCuda(hipGetDeviceProperties(&prop, 0));

	dim3 gridSize((n - 2) / (BLOCK_SIZE_x), (n - 2) / BLOCK_SIZE_y);
	dim3 blockSize(BLOCK_SIZE_x, BLOCK_SIZE_y);
	int blockmem = ((BLOCK_SIZE_y + 2) * (BLOCK_SIZE_x + 2) * (3 * sizeof(float) + 2 * sizeof(__int8)));
	//check if blocks fit in shared memory:
	if (blockmem > prop.sharedMemPerBlock) {
		throw "Block size too large!! \n";
	}

	if ((prop.maxThreadsPerMultiProcessor / (BLOCK_SIZE_x * BLOCK_SIZE_y)) * blockmem < 16 * pow(2, 10))
	{
		checkCuda(hipDeviceSetCacheConfig(hipFuncCachePreferL1));


	}

	if (n * n * (3 * sizeof(float) + 2 * sizeof(__int8)) > prop.totalGlobalMem)
	{
		throw "Device out of memory!! max size = %d ", sqrt(prop.totalGlobalMem / (3 * sizeof(float) + 2 * sizeof(__int8)));
	}

	float* H = initializeFloatArray();
	float* U = initializeFloatArray();
	float* V = initializeFloatArray();
	__int8* Upos = initializeBoolArray();
	__int8* Vpos = initializeBoolArray();

	copyConstants();


	fillarrays << <gridSize, blockSize >> > (H, Upos, Vpos);
	CudaCheckError();
	initializeWaterdrop(H);

	float* H_h = 0;
	float* V_h = 0;
	float* U_h = 0;
	hipHostMalloc(&H_h, n * n * sizeof(float));
	hipHostMalloc(&U_h, n * n * sizeof(float));
	hipHostMalloc(&V_h, n * n * sizeof(float));
	copyfromCudaMatrix(H_h, H, n, n);
	copyfromCudaMatrix(U_h, U, n, n);
	copyfromCudaMatrix(V_h, V, n, n);
	//showMatrix("H", H_h, n, n);
	unsigned numthreads;
	if (maxthreads>0)
	{
		numthreads = maxthreads;
	}
	else
	numthreads = std::thread::hardware_concurrency();
	if ((n-2) % numthreads != 0)
		while ((n-2) % numthreads != 0)
			--numthreads;

	std::thread* t = NULL;
	t = new std::thread[numthreads];
	auto cb = new cbar::cyclicbarrier(numthreads); //syncthreads(numthreads);	
	auto timestart = clock();
	for (int i = 0; i < numthreads; i++) {
	
		t[i] = std::thread(updatecputhreadborder, H_h,U_h,V_h,dt,i,numthreads,iter,cb);
	}

	for (int i = 0; i < numthreads; i++) {
		t[i].join();
	}
	float timestop = float(clock() - timestart)/ CLOCKS_PER_SEC;

	if (n == 34) {
		showMatrix("H", H_h, n, n);
	}
	printf("%.9f ",timestop);
	hipDeviceReset();
	

}

int main()
{	
	
	constexpr int ns[] = { 1  , 2, 4, 8, 16, 32};
	constexpr int iter = 10;
	constexpr int threads[] = { 0,1 };
		//int maxthreads = 1;
		for (const int ni : ns) {
			 n = 3 * 32 * ni + 2;
			
			//n = 34;
			
			
			printf("n = %d GPU: ", n);
			
			//runprogram(iter);
			runprogrambenchmark(iter,1);
			
			//printf("nobool: ");
			runprogrambenchmark(iter,2);
			
			//printf("borders: ");
			//runprogrambenchmark(iter,3);

			//RunProgramCPU(iter,0);
			//if (ni<20)
			printf("CPU: ");
			
			for (const int maxthreads:threads)
			RunProgramCPU(iter, maxthreads);
			
			printf(" \n");
		}
		hipDeviceReset();
	return 0;
}  